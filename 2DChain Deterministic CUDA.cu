#include "hip/hip_runtime.h"

/*  *** 2D CHAIN *** 
	Create Rigid Planar Chains on Honeycomb Lattice
	Check for Overlaps, Closed Loop Chains
	Systematic Approach, Checks All Chains of Given Length (less than 64 segments)
	GPU/CUDA Version, January 2014

	Rewritten to reflect number of segments rather than atoms, added minor improvements - March 2014

	By Christian Bracher */

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include <stdio.h>
#include <iostream>

/* Support for Timing */

#include <time.h> 

/* Check for 64-bit support (necessary) */

#if defined (_INTEGRAL_MAX_BITS) && \
  _INTEGRAL_MAX_BITS >= 64
typedef signed __int64 int64;
typedef unsigned __int64 uint64;
#else
#error __int64 type not supported
#endif

using namespace std;

/* *** GPU CONTROL PARAMETERS *** */

/* Threads per Block (default: 512, maximum: 1024) */
	int ThreadBits = 9;
	int ThreadsPerBlock = (int)(1 << ThreadBits);
	const int MaxThreadsPerBlock = 1024;

/* Number of Blocks (default: 4096)  */
	int BlockBits = 12;
	long NumberOfBlocks = ((long)1 << BlockBits);
	const long MaxNumberOfBlocks = 65536;

	
/* *** LATTICE VECTOR CLASS AND FUNCTIONS *** */

class LatticeVector
{
/* Grid Coordinates on 2D Triangular Grid */
public:
	int n1;
	int n2;

	__device__ LatticeVector ()
	{
	/* Default Constructor */
		n1 = 0;
		n2 = 0;
	}

	__device__ LatticeVector (int x, int y)
	{
	/* Create a Lattice Point */
		n1 = x;
		n2 = y;
	}

	__device__ void LeftTurn (int orientation)
	{
	/* Calculates Grid Position of New Terminal Atom for a Left Turn, Assuming Bond Orientation d */
		
		int d = orientation % 6;
		if (d < 0) d += 6;

		switch (d)
		{
		case 0:
			++n2;
			break;
		case 1:
			--n1;
			++n2;
			break;
		case 2:
			--n1;
			break;
		case 3:
			--n2;
			break;
		case 4:
			++n1;
			--n2;
			break;
		case 5:
			++n1;
			break;
		}
	}

	__device__ void RightTurn (int orientation)
	{
	/* Calculates Grid Position of New Terminal Atom for a Right Turn, Assuming Bond Orientation d */
		
		int d = orientation % 6;
		if (d < 0) d += 6;

		switch (d)
		{
		case 0:
			++n1;
			--n2;
			break;
		case 1:
			++n1;
			break;
		case 2:
			++n2;
			break;
		case 3:
			--n1;
			++n2;
			break;
		case 4:
			--n1;
			break;
		case 5:
			--n2;
			break;
		}
	}

	__device__ bool operator == (LatticeVector r1)
	{
	/* Compares Two Lattice Vectors */

		bool IsEqual = false;

		if ((r1.n1 == n1) && (r1.n2 == n2))  (IsEqual = true);

		return IsEqual;
	}
};

__device__ void BuildChain (uint64 Code, int length, LatticeVector *ChainArray)
{
/* Translate the Binary Code Into the Actual Lattice Points Occupied by the Chain */

	ChainArray[0] = LatticeVector(0,0);
	ChainArray[1] = LatticeVector(1,0);
	ChainArray[2] = LatticeVector(1,1);

	int orientation = 1;

	for (int k = 2; k < length; ++k)
	{
		ChainArray[k+1] = ChainArray[k];

		if (((Code>>(k-2)) % 2) == 0)
		{
			ChainArray[k+1].LeftTurn(orientation);
			++orientation;
		}
		else
		{
			ChainArray[k+1].RightTurn(orientation);
			--orientation;
		}
	}
}

__device__ bool IsChainOverlapping (int length, LatticeVector *ChainArray)
{
/* Compare All Pairs of "Atoms" for Occupying the Same Grid Position
   (note that it is impossible to form loops with less than six atoms,
   overlapping vertices are always an even number of segments apart) */

	for (int k1 = 0; k1 < length - 5 ; ++k1)
	{
		for (int k2 = k1 + 6; k2 <= length; k2 += 2)
		{
			if (ChainArray[k2]==ChainArray[k1]) return true;
		}	
	}

	return false;
}

__device__ bool IsChainClosedLoop (int length, LatticeVector *ChainArray)
{
/* For a Closed Loop, No Atoms Overlap Except the First and Last */

	if (!(ChainArray[length] == ChainArray[0]))
	{
	/* Check Ends of Chain First - Are They Different? */
		return false;
	}
	else
	{
	/* Check for Overlaps in the Interior of the Chain */
		for (int k1 = 0; k1 < length - 5 ; ++k1)
		{
			for (int k2 = k1 + 6; k2 <= length; k2 += 2)
			{
				if (ChainArray[k2]==ChainArray[k1])
				{
					if ((k1 > 0) || (k2 < length))
						return false;
				}
			}	
		}
	}
	return true;
}

/* *** The ChainCounts Structure *** */
	
struct ChainCounts
	{
		unsigned long NonOverlapping;
		unsigned long ClosedChains;
	};

/* NEEDS TO BE ADAPTED... */ 

void PrintChainArray (int length, LatticeVector* ChainArray)
{
/* Print the Coordinates of the Atomic Grid Sites in the Chain */

	for(int i=0; i <= length; ++i)
	{
		cout << "(" << ChainArray[i].n1 << "," << ChainArray[i].n2 << ") ";
	}
	cout << "\n";
}

/* *** Parallel Code for Chain Analysis *** */

__global__ void CUDAChainAnalyze(ChainCounts *CUDAChainInfo, uint64 CodeOffset, int ChainLength)
{
/* Prepare Block Cache For Overlap Data */
	__shared__ ChainCounts DataCache[MaxThreadsPerBlock];

/* Allocate Memory for Building Chains (maximum length: 64) */
	LatticeVector MyChainArray[64];

/* Figure Out Correct Code For Two-Dimensional Chain */
	uint64 MyCode = CodeOffset + (uint64)(threadIdx.x + blockIdx.x * blockDim.x);	

/* Build the Chain ... */
	BuildChain(MyCode, ChainLength, MyChainArray);

/* ... and Test It For Overlaps (default value: overlaps): */
	DataCache[threadIdx.x].NonOverlapping = 0;
	
	if (IsChainOverlapping(ChainLength, MyChainArray) == false)
		DataCache[threadIdx.x].NonOverlapping = 1;
		
/* ... and Test It For Closed Chains (default value: not closed): */
	DataCache[threadIdx.x].ClosedChains = 0;
	
	if (IsChainClosedLoop(ChainLength, MyChainArray) == true)
		DataCache[threadIdx.x].ClosedChains = 1;
		
/* Wait for Tests Within a Block To Be Completed */
	__syncthreads();

/* Now, Add Results Within Block */
	int AddLimit = blockDim.x / 2;
	
	while (AddLimit > 0)
	{
		if (threadIdx.x < AddLimit)
		{
			DataCache[threadIdx.x].NonOverlapping += DataCache[threadIdx.x + AddLimit].NonOverlapping;
			DataCache[threadIdx.x].ClosedChains   += DataCache[threadIdx.x + AddLimit].ClosedChains;
		}

		__syncthreads();

		AddLimit /= 2;
	}

/* Store Result (now in position 0) in Global Memory */
	if (threadIdx.x == 0)
	{
		CUDAChainInfo[blockIdx.x].NonOverlapping = DataCache[0].NonOverlapping;
		CUDAChainInfo[blockIdx.x].ClosedChains = DataCache[0].ClosedChains;
	}
}

/* *** TIMING FUNCTONS *** */

double Duration (clock_t initial, clock_t final)
{
/* Determine the Duration of a Calculation in Seconds */

	double TimeDiff = (double)(final - initial);
	return TimeDiff / CLOCKS_PER_SEC;
}

/* *** MAIN PROGRAM STARTS HERE *** */

int main()
{
	int Length;
	clock_t StartTime, FinishTime;
	double StartToFinish;

/* Enter Chain Length Examined */
	cout << "Enter Chain Length: ";
	cin >> Length;
	cout << "\n\n";

/* Timing Support - Start of Calculation */
	StartTime = clock();

//	hipProfilerStart();
		
/* Message */
	cout << "Calculating chains of length " << Length << " : ";

/* There are 2^(l-2) Different Chains */ 
	uint64 MaxCode = ((uint64)1 << (Length - 2));

/* Chop GPU Calculation Into Pieces If Length Exceeds ThreadBits + BlockBits + 2 */
	uint64 CUDAIterations;
	
	if (MaxCode > (NumberOfBlocks * ThreadsPerBlock))
	{
	/* Number of Global Iterations */
		CUDAIterations = ((MaxCode >> BlockBits) >> ThreadBits);
	}
	else
	{
	/* Adjust Number of Blocks */
		CUDAIterations = 1;
		NumberOfBlocks = (long)(MaxCode >> ThreadBits);

	/* Check: There Must Be At Least As Many Chains As Threads */
		if (NumberOfBlocks == 0)
		{
			cerr << "ERROR:  Insufficient Length of Chain \n\n";
			exit(1);
		}
	}
	
/* Initialize Variable to Save Numbers of Non-Overlapping Chains, Closed Chains */
	uint64 NonOverlapping = 0;
	uint64 ClosedChains = 0;

/* Reserve Memory to Transfer Information Between CPU and GPU */
	ChainCounts *CPUChainInfo = new ChainCounts[NumberOfBlocks];

	ChainCounts *CUDAChainInfo; 
	hipError_t cudaStatus = hipMalloc((void**)&CUDAChainInfo, NumberOfBlocks * sizeof(ChainCounts));

/* Loop Through All Possible Configurations */
	for (uint64 Iter = 0; Iter < CUDAIterations; ++Iter)
	{
	/* Examine (ThreadsPerBlock * BlockNumber) Chains in Parallel */

	/* Determine Offset for Parallel Calculation */
		uint64 Offset = Iter * NumberOfBlocks * ThreadsPerBlock;

	/* Perform Parallel Analysis */
		CUDAChainAnalyze <<<NumberOfBlocks,ThreadsPerBlock>>> (CUDAChainInfo, Offset, Length);

	/* Copy Results to Host Memory */
		cudaStatus = hipMemcpy(CPUChainInfo, CUDAChainInfo, NumberOfBlocks * sizeof(ChainCounts), hipMemcpyDeviceToHost);

	/* Extract Information */
		for (uint64 BlockID = 0; BlockID < NumberOfBlocks; ++BlockID)
		{
			NonOverlapping += (uint64)CPUChainInfo[BlockID].NonOverlapping;
			ClosedChains   += (uint64)CPUChainInfo[BlockID].ClosedChains; 
		}

	/* Indicate Progress */
		cout << ".";
	}

	hipDeviceReset();

/* Timing Support - End of Calculation */
	FinishTime = clock();
	StartToFinish = Duration(StartTime, FinishTime);

/* Send a Brief Message */
		cout << " done!\n\n"
			 << "Found " << NonOverlapping << " non-overlapping chains.\n"
			 << "Found " << ClosedChains << " closed chains \n\n"
			 << "Time of Calculation: " << StartToFinish << " seconds.\n\n";

/* Wait for Key: */
	char Aux;
	cout << "Hit A Key, Then ENTER\n\n";
	cin >> Aux; 

/* Cleanup & Done! */
	delete[] CPUChainInfo;
	hipFree(CUDAChainInfo);

	return 0;
}